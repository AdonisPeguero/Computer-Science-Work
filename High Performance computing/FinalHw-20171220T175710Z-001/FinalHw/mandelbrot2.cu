#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <gd.h>
#include <gdfontl.h>
#include <math.h>

void drawImage(double * buffer,int n,const char * fname); 

double XMIN=-2.0;
double XMAX=2.0;
double YMIN=-2.0;
double YMAX=2.0;
int COUNT_MAX=1000;

__global__ void pixelValue(int n, double XMIN, double XMAX, double YMIN, double YMAX, int COUNT_MAX, double * img_buffer) {
    double cx, cy;
    double tx, ty;
    double zx, zy;
    double img_val;
    int count;
    double BOUNDARY=5.0;
    int ij,i,j;

    //   ADD COMMAND HERE TO SET VALUE OF IJ, GLOBAL THREAD INDEX
    ij = threadIdx.x + blockIdx.x*blockDim.x;

    while(ij<n*n) {
        i = ij/n;
        j = ij%n;
    
        cx = XMIN + (double)i/(double)n*(XMAX-XMIN);
        cy = YMAX - (double)j/(double)n*(YMAX-YMIN);
        zx = 0.0;
        zy = 0.0;
    
        count=0;
        img_val=0.0;
        while(count<COUNT_MAX) {
            count++;
            tx = zx*zx-zy*zy+cx;
            ty = 2*zx*zy+cy;
            zx = tx;    
            zy = ty;
            if((zx*zx+zy*zy)>BOUNDARY) {
                img_val= log((double)count);
                break;
            }
        }
        img_buffer[ij]=img_val;

        // ADD COMMAND HERE TO INCREMENT IJ, BY THE GLOBAL SIZE
		ij += blockDim.x*gridDim.x;
    }
}

int main(int argc, char ** argv) {
    int i,j,ij,n;
    double * img_buffer;
    // DEFINE dev_img_buffer
	double * dev_img_buffer;
    hipEvent_t start,stop;
    float time_elapsed=0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    n=100;
    i=1;
    if(argc>i) sscanf(argv[i++],"%d",&n);
    if(argc>i) sscanf(argv[i++],"%lf",&XMIN);
    if(argc>i) sscanf(argv[i++],"%lf",&XMAX);
    if(argc>i) sscanf(argv[i++],"%lf",&YMIN);
    if(argc>i) sscanf(argv[i++],"%lf",&YMAX);
    if(argc>i) sscanf(argv[i++],"%d",&COUNT_MAX);


    printf("Mandelbrot Example\n");
    printf("N = %d\n",n);
    printf("X = %lf -> %lf, Y = %lf -> %lf\n",XMIN,XMAX,YMIN,YMAX);
    printf("DEPTH = %d\n",COUNT_MAX);

    img_buffer = (double *)malloc(sizeof(double)*n*n);
    // ALLOCATE dev_img_buffer
	hipMalloc((void **)&dev_img_buffer,sizeof(double)*n*n);

    hipEventRecord(start);
    // CALL pixelValue with 39 blocks and 64 threads per block 
	pixelValue<<<39,64>>>(n, XMIN, XMAX, YMIN, YMAX, COUNT_MAX,dev_img_buffer);
    // COPY dev_img_buffer to img_buffer
	hipMemcpy(img_buffer,dev_img_buffer,sizeof(double)*n*n,hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed,start,stop);

    printf("ELAPSED TIME IN CALCULATION LOOP = %lf ms\n",time_elapsed);

    printf("Drawing image\n");
    drawImage(img_buffer,n,"file.png");

    free(img_buffer);
    // FREE dev_img_buffer
	hipFree((void*)dev_img_buffer);
    return 0;

}

gdImagePtr im;
FILE *pngout;
int black;
int white;
#define NCOLORS 200
int cmap[NCOLORS];

void drawImage(double * buffer,int n,const char * fname) {
    int i,j,scale;
    double max,min,ncolors;
    im = gdImageCreate(n,n);
    black = gdImageColorResolve(im, 0, 0, 0);
    white = gdImageColorResolve(im, 255, 255, 255);
    ncolors = (double)NCOLORS;
    for (i=0;i<NCOLORS;i++) {
        cmap[i] = gdImageColorResolve(im,
            (int)((ncolors-i)/ncolors*255.0),
            (int)((ncolors/2.0-abs(ncolors/2.0-i))/(ncolors/2.0)*255.0),
            (int)((i)/ncolors*255.0)
        );
    }
    cmap[0] = black;

    gdImageFilledRectangle(im,0,0,n,n,white);
    max=buffer[0];
    min=buffer[0];
    for(i=0;i<n*n;i++) {
        if(buffer[i]>max) max=buffer[i];
        if(buffer[i]<min) min=buffer[i];
    }
    if(max==min) { max += 0.5; min -= 0.5; }
    for(i=0;i<n;i++) {
        for(j=0;j<n;j++) {
            scale = (int)((buffer[i*n+j]-min)/(max-min)*ncolors);
            if(scale<0) scale=0;
            if(scale>99) scale=99;
            gdImageSetPixel(im,i,j,cmap[scale]);
        }
    }
    gdImageRectangle(im,0,0,n-1,n-1,black);
    pngout = fopen(fname, "wb");
    gdImagePng(im, pngout);
    fclose(pngout);

    //cleanup
    gdImageDestroy(im);
}


